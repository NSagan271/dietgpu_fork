#include "hip/hip_runtime.h"
#include "dietgpu/ans/BatchProvider.cuh"
#include "dietgpu/float/GpuFloatCodec.h"
#include "dietgpu/float/GpuSparseFloatCompress.cuh"
#include "dietgpu/float/GpuFloatUtils.cuh"
#include "dietgpu/utils/DeviceUtils.h"
#include "dietgpu/utils/StackDeviceMemory.h"
#include "dietgpu/utils/StaticUtils.h"

#include <glog/logging.h>
#include <cmath>
#include <memory>
#include <vector>

namespace dietgpu {

uint32_t getMaxSparseFloatCompressedSize(FloatType floatType, uint32_t size) {
  // The maximum number of compressed bytes (in the worst case of 100% dense data)
  // is the value output getMaxFloatCompressedSize, plus the number of bytes needed
  // to store the bitmap and the sparse float header.
  uint32_t bitmapSize = roundUp((size + 7) / 8, 16);
  uint32_t baseSize = sizeof(GpuSparseFloatHeader);

  return baseSize + bitmapSize + getMaxFloatCompressedSize(floatType, size);
}

/* Performs compression on sparse floats. This has the same API as floatCompress,
 * but uses a specialized algorithm for compressing sparse floats: first, it
 * generates a bitmap describing whether each element of the input dataset is
 * zero or nonzero. Then, it performs regular float compression, only on the
 * nonzero elements.
 */
void floatCompressSparse(
    StackDeviceMemory& res,
    const FloatCompressConfig& config,
    uint32_t numInBatch,
    const void** in,
    const uint32_t* inSize,
    void** out,
    uint32_t* outSize_dev,
    hipStream_t stream) {
  // Get the total and maximum input size
  uint32_t maxSize = 0;

  for (uint32_t i = 0; i < numInBatch; ++i) {
    maxSize = std::max(maxSize, inSize[i]);
  }

  // Copy data to device
  // To reduce latency, we prefer to coalesce all data together and copy as one
  // contiguous chunk
  static_assert(sizeof(void*) == sizeof(uintptr_t), "");
  static_assert(sizeof(uint32_t) <= sizeof(uintptr_t), "");

  // in, inSize, out
  auto params_dev = res.alloc<uintptr_t>(stream, numInBatch * 3);
  auto params_host =
      std::unique_ptr<uintptr_t[]>(new uintptr_t[3 * numInBatch]);

  std::memcpy(&params_host[0], in, numInBatch * sizeof(void*));
  std::memcpy(&params_host[numInBatch], inSize, numInBatch * sizeof(uint32_t));
  std::memcpy(&params_host[2 * numInBatch], out, numInBatch * sizeof(void*));

  CUDA_VERIFY(hipMemcpyAsync(
      params_dev.data(),
      params_host.get(),
      3 * numInBatch * sizeof(uintptr_t),
      hipMemcpyHostToDevice,
      stream));

  auto in_dev = (const void**)params_dev.data();
  auto inSize_dev = (const uint32_t*)(params_dev.data() + numInBatch);
  auto out_dev = (void**)(params_dev.data() + 2 * numInBatch);

  auto inProvider = BatchProviderPointer((void**)in_dev, inSize_dev);
  auto outProvider = BatchProviderPointer(out_dev);

  floatCompressSparseDevice(
      res,
      config,
      numInBatch,
      inProvider,
      maxSize,
      outProvider,
      outSize_dev,
      stream);
}

} // namespace dietgpu
